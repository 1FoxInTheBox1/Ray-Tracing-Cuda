#include "hip/hip_runtime.h"
#include "../include/color.cuh"
#include "../include/ray.cuh"
#include "../include/vec3.cuh"

#include <iostream>

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ color ray_color(const ray &r)
{
    vec3 unit_direction = unit_vector(r.direction());
    auto a = 0.5*(unit_direction.y() + 1.0);
    return (1.0-a)*color(1.0, 1.0, 1.0) + a*color(0.5, 0.7, 1.0);
}

__global__ void render(color *fb, int image_width, int image_height, vec3 pixel00_loc, vec3 pixel_delta_u, vec3 pixel_delta_v, vec3 camera_center)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= image_width || j >= image_height)
        return;

    int pixel_index = i + image_width * j;
    auto pixel_center = pixel00_loc + (i * pixel_delta_u) + (j * pixel_delta_v);
    auto ray_direction = pixel_center - camera_center;
    ray r(camera_center, ray_direction);

    color pixel_color = ray_color(r);

    fb[pixel_index] = pixel_color;
}

int main()
{
    // Image
    int image_width = 400;
    auto aspect_ratio = 16.0 / 9.0;
    int image_height = int(image_width / aspect_ratio);
    image_height = (image_height < 1) ? 1 : image_height;

    int tx = 8;
    int ty = 8;
    int num_pixels = image_height * image_width;
    size_t fb_size = 3 * num_pixels * sizeof(color);

    color *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // Camera
    auto focal_length = 1.0f;
    auto viewport_height = 2.0f;
    auto viewport_width = viewport_height * (double(image_width) / image_height);
    auto camera_center = point3(0, 0, 0);

    // Calculate the vectors across the horizontal and down the vertical viewport edges.
    auto viewport_u = vec3(viewport_width, 0, 0);
    auto viewport_v = vec3(0, -viewport_height, 0);

    // Calculate the horizontal and vertical delta vectors from pixel to pixel.
    auto pixel_delta_u = viewport_u / image_width;
    auto pixel_delta_v = viewport_v / image_height;

    // Calculate the location of the upper left pixel.
    auto viewport_upper_left = camera_center - vec3(0, 0, focal_length) - viewport_u / 2 - viewport_v / 2;
    auto pixel00_loc = viewport_upper_left + 0.5f * (pixel_delta_u + pixel_delta_v);

    // Render
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(fb, image_width, image_height, pixel00_loc, pixel_delta_u, pixel_delta_v, camera_center);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n"
              << image_width << ' ' << image_height << "\n255\n";

    for (int j = 0; j < image_height; j++)
    {
        for (int i = 0; i < image_width; i++)
        {
            int pixel_index = i + image_width * j;
            color pixel_color = fb[pixel_index];
            write_color(std::cout, pixel_color);
        }
    }

    std::clog << "\r Done.           \n";

    checkCudaErrors(hipFree(fb));
}